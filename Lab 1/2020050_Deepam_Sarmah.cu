/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>


#define LENGTH 100000000
using namespace std;

__global__ void vector_add_gpu (float *a, float *b, float *c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

void vector_add_cpu(float *a, float *b, float *c){
    for (int i = 0; i < LENGTH; i ++){
        c[i] = a[i] + b[i];
    }
}

int main(){

    float *a_vec;
    float *b_vec;
    
    float *c_vec;
    float *d_a, *d_b, *d_c;
    float *h_c;


    h_c = (float*)malloc(LENGTH*sizeof(float));
    a_vec = (float*)malloc(LENGTH*sizeof(float));
    b_vec = (float*)malloc(LENGTH*sizeof(float));

    for (int i = 0; i < LENGTH; i ++){
        a_vec[i] = i;
        b_vec[i] = i;
    }

    timespec begin, end;

    clock_gettime(CLOCK_REALTIME, &begin);
    vector_add_cpu(a_vec, b_vec, h_c);
    clock_gettime(CLOCK_REALTIME, &end);

    long seconds = end.tv_sec - begin.tv_sec;
    long nanoseconds = end.tv_nsec - begin.tv_nsec;

    double elapsed = seconds + nanoseconds*1e-9;

    // for(int i=0 ; i< LENGTH/100000 ; i++){
	// 	std::cout << h_c[i] << std::endl;
	// }

    printf("[CPU] Time measured: %.3f seconds.\n", elapsed);

    // -- GPU CODE --
    hipMalloc((void **) &d_a, LENGTH * sizeof(float));
    hipMalloc((void **) &d_b, LENGTH * sizeof(float));
    hipMalloc((void **) &d_c, LENGTH * sizeof(float));
    
    hipMemcpy(d_a, a_vec, LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_vec, LENGTH * sizeof(float), hipMemcpyHostToDevice);
    
    float gpu_elapsed = 0.0f;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);
    
    vector_add_gpu<<< 1 + LENGTH / 256, 256>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, LENGTH * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(gpu_stop); 
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    printf("[GPU] Time measured: %.10f milliseconds.\n", gpu_elapsed);
    return 0;
}