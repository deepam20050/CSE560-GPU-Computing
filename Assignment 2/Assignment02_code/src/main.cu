#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>

#define kernelWidth 3
#define kernelHeight 3
#define imageChannels 1

using namespace std;

__constant__ float imageKernel_c[kernelHeight * kernelWidth];

void sequentialConvolution(const unsigned char*inputImageData, const float *kernel ,unsigned char * outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels)
{
	int i, j, m, n, mm, nn;
	int kCenterX, kCenterY;
	float sum;
	int yIndex, xIndex;

	kCenterX = kernelSizeX / 2;
	kCenterY = kernelSizeY / 2;

	for (int k=0; k<channels; k++) 
	{
		for (i = 0; i < dataSizeY; ++i)
		{
			for (j = 0; j < dataSizeX; ++j)
			{
				sum = 0;
				for (m = 0; m < kernelSizeY; ++m)
				{
					mm = kernelSizeY - 1 - m;

					for (n = 0; n < kernelSizeX; ++n)
					{
						nn = kernelSizeX - 1 - n;

						yIndex = i + m - kCenterY;
						xIndex = j + n - kCenterX;

						if (yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX)
							sum += inputImageData[(dataSizeX * yIndex + xIndex)*channels + k] * kernel[kernelSizeX * mm + nn];
					}
				}
				outputImageData[(dataSizeX * i + j)*channels + k] = sum;
			}
		}
	}
}

__global__ void gpu1 (const unsigned char * InputImageData, const float * kernel, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < dataSizeY && j < dataSizeX) {
		int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
		for (int k = 0; k < channels; ++k) {
			float sum = 0.0f;
			for (int m = 0; m < kernelSizeY; ++m) {
				int mm = kernelSizeY - 1 - m;
				for (int n = 0; n < kernelSizeX; ++n) {
					int nn = kernelSizeX - 1 - n;
					int yIndex = i + m - kCenterY;
					int xIndex = j + n - kCenterX;
					if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
						sum += InputImageData[(yIndex * dataSizeX + xIndex) * channels + k] * kernel[kernelSizeX * mm + nn];
					}
				}
			}
			outputImageData[(i * dataSizeX + j) * channels + k] = sum;
		}
	}
}

__global__ void gpu2 (const unsigned char * InputImageData, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
  __shared__ unsigned char tile[32][32][imageChannels];
  for (int k = 0; k < channels; ++k) {
    for (int m = 0; m < kernelSizeY; ++m) {
      for (int n = 0; n < kernelSizeX; ++n) {
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if( yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          tile[ty + m][tx + n][k] = InputImageData[(yIndex * dataSizeX + xIndex) * channels + k];
        } else {
          tile[ty + m][tx + n][k] = 0;
        }
      }
    }
  }
  __syncthreads();
  for (int k = 0; k < channels; ++k) {
    float sum = 0.0f;
    for (int m = 0; m < kernelSizeY; ++m) {
      int mm = kernelSizeY - 1 - m;
      for (int n = 0; n < kernelSizeX; ++n) {
        int nn = kernelSizeX - 1 - n;
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          sum += tile[ty + m][tx + n][k] * imageKernel_c[kernelSizeX * mm + nn];
        }
      }
    }
    outputImageData[(i * dataSizeX + j) * channels + k] = sum;
  }
}

int main(int argc, char* argv[]){
	if(argc < 3) {
		cout<<"Usage: "<<argv[0]<<" <image_in> <image_out>\n";
		return 0;
	}
	// Read input image on host
	int imageWidth, imageHeight, bpp;
	const unsigned char* image_in = stbi_load( argv[1], &imageWidth, &imageHeight, &bpp, imageChannels );
	if(bpp != 1 || imageChannels != 1) {
		cout<<"Input image must be 8 bits per pixel, and sigle channel (grayscale).\n";
		return 0;
	}
	cout << "Image size: " << imageHeight << " x " << imageWidth << std::endl; 
	
	// Allocate output image memory on host
	unsigned char *image_out = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));
	unsigned char *image_gpu1 = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));
	unsigned char *image_gpu2 = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));

	// Setup image convolution kernel on host
	float imageKernel[kernelHeight*kernelWidth];
	for(int i=0; i< kernelWidth*kernelHeight; i++){
		imageKernel[i] = 1.0/(kernelHeight*kernelWidth);
	}

	// Perform image convolution
	sequentialConvolution(image_in, imageKernel, image_out, kernelWidth, 
	kernelHeight, imageWidth, imageHeight, imageChannels);

	// Write convolved image to disk
	stbi_write_png(argv[2], imageWidth, imageHeight, imageChannels, image_out, 0);

	// Add cuda code here
	
	// allocate memory on device
	int size = imageWidth * imageHeight;
	unsigned char *device_image_gpu1, *device_image_gpu2, *image_in_gpu;
	float *imageKernel_gpu;
	hipMalloc((void **)&device_image_gpu1, size * sizeof(unsigned char));
	hipMalloc((void **)&device_image_gpu2, size * sizeof(unsigned char));
	hipMalloc((void **)&image_in_gpu, size * sizeof(unsigned char));
	hipMalloc((void **)&imageKernel_gpu, kernelHeight * kernelWidth * sizeof(float));
	hipMemcpy(image_in_gpu, image_in, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(imageKernel_gpu, imageKernel, kernelHeight * kernelWidth * sizeof(float), hipMemcpyHostToDevice);

	// creating blocks and grids
	dim3 block(16, 16);
	dim3 grid((imageHeight + block.x - 1) / block.x, (imageWidth + block.y - 1) / block.y);

	// launching GPU1 kernel
	// TODO : Timing GPU1 Kernel
	gpu1<<<grid, block>>>(image_in_gpu, imageKernel_gpu, device_image_gpu1, kernelWidth, kernelHeight, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();

	hipMemcpy(image_gpu1, device_image_gpu1, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	string gpu1_png(argv[2]);
	gpu1_png.pop_back(); gpu1_png.pop_back(); gpu1_png.pop_back(); gpu1_png.pop_back();
	gpu1_png += "-GPU1.png";
	stbi_write_png(gpu1_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu1, 0);

	// copying imageKernel to imageKernel_c(constant memory)
	hipMemcpyToSymbol(HIP_SYMBOL(imageKernel_c), imageKernel, kernelHeight * kernelWidth * sizeof(float));

	// launching GPU2 kernel
	// TODO : Timing GPU2 Kernel
	gpu2<<<grid, block>>>(image_in_gpu, device_image_gpu2, kernelWidth, kernelHeight, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();

	hipMemcpy(image_gpu2, device_image_gpu2, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	string gpu2_png(argv[2]);
	gpu2_png.pop_back(); gpu2_png.pop_back(); gpu2_png.pop_back(); gpu2_png.pop_back();
	gpu2_png += "-GPU2.png";
	stbi_write_png(gpu2_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu2, 0);

	//Deallocate memory
	free(image_out);
	free(image_gpu1);
	free(image_gpu2);
	hipFree(device_image_gpu1);
	hipFree(device_image_gpu2);
	hipFree(image_in_gpu);
	hipFree(imageKernel_gpu);
	return 0;
}
