#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>

#define kernelWidth 3
#define kernelHeight 3
#define imageChannels 1

using namespace std;

void sequentialConvolution(const unsigned char*inputImageData, const float *kernel ,unsigned char * outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels)
{
	int i, j, m, n, mm, nn;
	int kCenterX, kCenterY;
	float sum;
	int yIndex, xIndex;

	kCenterX = kernelSizeX / 2;
	kCenterY = kernelSizeY / 2;

	for (int k=0; k<channels; k++) 
	{
		for (i = 0; i < dataSizeY; ++i)
		{
			for (j = 0; j < dataSizeX; ++j)
			{
				sum = 0;
				for (m = 0; m < kernelSizeY; ++m)
				{
					mm = kernelSizeY - 1 - m;

					for (n = 0; n < kernelSizeX; ++n)
					{
						nn = kernelSizeX - 1 - n;

						yIndex = i + m - kCenterY;
						xIndex = j + n - kCenterX;

						if (yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX)
							sum += inputImageData[(dataSizeX * yIndex + xIndex)*channels + k] * kernel[kernelSizeX * mm + nn];
					}
				}
				outputImageData[(dataSizeX * i + j)*channels + k] = sum;
			}
		}
	}
}

__global__ void gpu1 (const unsigned char * InputImageData, const float * kernel, unsigned char* outputImageData, int imageWidth, int imageHeight, int channels) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < imageHeight && j < imageWidth) {
		int kCenterX = kernelWidth / 2, kCenterY = kernelHeight / 2;
		for (int k = 0; k < channels; ++k) {
			float sum = 0.0f;
			for (int m = 0; m < kernelHeight; ++m) {
				int mm = kernelHeight - 1 - m;
				for (int n = 0; n < kernelWidth; ++n) {
					int nn = kernelWidth - 1 - n;
					int yIndex = i + m - kCenterY;
					int xIndex = j + n - kCenterX;
					if(yIndex >= 0 && yIndex < imageHeight && xIndex >= 0 && xIndex < imageWidth){
						sum += InputImageData[(yIndex * imageWidth + xIndex) * channels + k] * kernel[kernelWidth * mm + nn];
					}
				}
			}
			outputImageData[(i * imageWidth + j) * channels + k] = sum;
		}
	}
}

int main(int argc, char* argv[]){
	if(argc < 3) {
		cout<<"Usage: "<<argv[0]<<" <image_in> <image_out>\n";
		return 0;
	}

	// Read input image on host
	int imageWidth, imageHeight, bpp;
	const unsigned char* image_in = stbi_load( argv[1], &imageWidth, &imageHeight, &bpp, imageChannels );
	if(bpp != 1 || imageChannels != 1) {
		cout<<"Input image must be 8 bits per pixel, and sigle channel (grayscale).\n";
		return 0;
	}
	cout << "Image size: " << imageHeight << " x " << imageWidth << std::endl; 

	// Allocate output image memory on host
	unsigned char *image_out = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));


	// Setup image convolution kernel on host
	float imageKernel[kernelHeight*kernelWidth];
	for(int i=0; i< kernelWidth*kernelHeight; i++){
		imageKernel[i] = 1.0/(kernelHeight*kernelWidth);
	}

	// Perform image convolution
	sequentialConvolution(image_in, imageKernel, image_out, kernelWidth, 
	kernelHeight, imageWidth, imageHeight, imageChannels);

	// Write convolved image to disk
	stbi_write_png(argv[2], imageWidth, imageHeight, imageChannels, image_out, 0);


	// Add cuda code here
	// creating blocks and grids + allocating memory
	int size = imageWidth * imageHeight;
	unsigned char *device_image_gpu1;
	unsigned char *image_gpu1 = new unsigned char[size];
	unsigned char *image_gpu2 = new unsigned char[size];
	hipMalloc((void **)&device_image_gpu1, size * sizeof(unsigned char));
	dim3 block(16, 16);
	dim3 grid((imageHeight + block.x - 1) / block.x, (imageWidth + block.y - 1) / block.y);
	
	// launching GPU1 kernel
	// TODO : Timing GPU1 Kernel
	gpu1<<<grid, block>>>(image_in, imageKernel, device_image_gpu1, kernelWidth, 
												kernelHeight, imageChannels);
	hipDeviceSynchronize();

	hipMemcpy(image_gpu1, device_image_gpu1, size, hipMemcpyDeviceToHost);
	string gpu1_png(argv[2]);
	gpu1_png.pop_back();
	gpu1_png.pop_back();
	gpu1_png.pop_back();
	gpu1_png.pop_back();
	gpu1_png += "-GPU1.png";
	stbi_write_png(gpu1_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu1, 0);


	// launching GPU2 kernel
	// TODO : Writing GPU2 + Timing GPU2 Kernel

	//Deallocate memory
	free(image_out);
	delete[] image_gpu1;
	delete[] image_gpu2;
	hipFree(device_image_gpu1);
	return 0;
}
