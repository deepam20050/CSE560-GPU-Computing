
#include <hip/hip_runtime.h>
__global__ void convKernel(unsigned char * inputImageData, const float * kernel,
        unsigned char* outputImageData, int channels, int imageWidth, int imageHeight, int kernelWidth, int kernelHeight){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < imageHeight && j < imageWidth){
        for(int k=0; k<channels; k++){
            float sum = 0.0;
            int kCenterX = kernelWidth/2;
            int kCenterY = kernelHeight/2;

            for(int m=0; m<kernelHeight; m++){
                int mm = kernelHeight - 1 - m;
                for(int n=0; n<kernelWidth; n++){
                    int nn = kernelWidth - 1 - n;

                    int yIndex = i + m - kCenterY;
                    int xIndex = j + n - kCenterX;

                    if(yIndex >= 0 && yIndex < imageHeight && xIndex >= 0 && xIndex < imageWidth){
                        sum += (float)inputImageData[(yIndex*imageWidth + xIndex)*channels + k] * kernel[mm*kernelWidth + nn];
                    }
                }
            }
            outputImageData[(i*imageWidth + j)*channels + k] = (unsigned char)sum;
        }
    }
}
