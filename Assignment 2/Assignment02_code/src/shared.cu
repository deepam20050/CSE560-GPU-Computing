#include "hip/hip_runtime.h"
__global__ void convKernel(unsigned char* inputImageData, unsigned char* outputImageData, int channels, int imageWidth, int imageHeight)
{
    __shared__ unsigned char tile[imageChannels][32][32];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * 30;
    int by = blockIdx.y * 30;
    int x = bx + tx;
    int y = by + ty;

    // Load input tile from global memory to shared memory
    for (int k = 0; k < channels; k++)
    {
        if (x < imageWidth && y < imageHeight)
            tile[k][ty][tx] = inputImageData[(y * imageWidth + x) * channels + k];
        else
            tile[k][ty][tx] = 0;
    }

    __syncthreads();

    float sum = 0;
    int kCenterX = kernelWidth / 2;
    int kCenter