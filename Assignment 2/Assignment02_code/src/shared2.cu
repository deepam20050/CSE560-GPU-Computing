#include "hip/hip_runtime.h"
__global__ void convKernel(const unsigned char * inputImageData, const float * __restrict__ kernel,
                           unsigned char* outputImageData, int channels, int imageWidth, int imageHeight,
                           int kernelSizeX, int kernelSizeY)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int blockWidth = blockDim.x;
    const int blockHeight = blockDim.y;
    const int imageChannels = channels;

    __shared__ unsigned char inputImageTile[(blockWidth + kernelSizeX - 1)][(blockHeight + kernelSizeY - 1)][imageChannels];
    
    const int kCenterX = kernelSizeX / 2;
    const int kCenterY = kernelSizeY / 2;
    
    const int x = bx * blockWidth + tx - kCenterX;
    const int y = by * blockHeight + ty - kCenterY;

    float sum = 0;
    
    if(x >= 0 && y >= 0 && x < imageWidth && y < imageHeight) {
        for(int k = 0; k < imageChannels; ++k) {
            inputImageTile[ty][tx][k] = inputImageData[(y * imageWidth + x) * imageChannels + k];
        }
    } else {
        for(int k = 0; k < imageChannels; ++k) {
            inputImageTile[ty][tx][k] = 0;
        }
    }

    __syncthreads();

    if(tx < blockWidth && ty < blockHeight && x < imageWidth && y < imageHeight) {
        for (int m = 0; m < kernelSizeY; ++m) {
            for (int n = 0; n < kernelSizeX; ++n) {
                const int mm = kernelSizeY - 1 - m;
                const int nn = kernelSizeX - 1 - n;
                const int yy = ty + m;
                const int xx = tx + n;

                for(int k = 0; k < imageChannels; ++k) {
                    sum += inputImageTile[yy][xx][k] * kernel[mm * kernelSizeX + nn];
                }
            }
        }
        for(int k = 0; k < imageChannels; ++k) {
            outputImageData[(y * imageWidth + x) * imageChannels + k] = sum;
        }
    }
}
