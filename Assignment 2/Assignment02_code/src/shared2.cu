#include "hip/hip_runtime.h"
__global__ void gpu2 (const unsigned char * InputImageData, const float * kernel, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int blockWidth = blockDim.x;
    const int blockHeight = blockDim.y;

    __shared__ unsigned char inputImageTile[(blockWidth + kernelSizeX - 1)][(blockHeight + kernelSizeY - 1)][channels];
    
    const int kCenterX = kernelSizeX / 2;
    const int kCenterY = kernelSizeY / 2;
    
    const int x = bx * blockWidth + tx - kCenterX;
    const int y = by * blockHeight + ty - kCenterY;

    float sum = 0;
    
    if(x >= 0 && y >= 0 && x < dataSizeX && y < dataSizeY) {
        for(int k = 0; k < channels; ++k) {
            inputImageTile[ty][tx][k] = InputImageData[(y * dataSizeX + x) * channels + k];
        }
    } else {
        for(int k = 0; k < channels; ++k) {
            inputImageTile[ty][tx][k] = 0;
        }
    }

    __syncthreads();

    if(tx < blockWidth && ty < blockHeight && x < dataSizeX && y < dataSizeY) {
        for (int m = 0; m < kernelSizeY; ++m) {
            for (int n = 0; n < kernelSizeX; ++n) {
                const int mm = kernelSizeY - 1 - m;
                const int nn = kernelSizeX - 1 - n;
                const int yy = ty + m;
                const int xx = tx + n;

                for(int k = 0; k < channels; ++k) {
                    sum += inputImageTile[yy][xx][k] * kernel[mm * kernelSizeX + nn];
                }
            }
        }
        for(int k = 0; k < channels; ++k) {
            outputImageData[(y * dataSizeX + x) * channels + k] = sum;
        }
    }
}
