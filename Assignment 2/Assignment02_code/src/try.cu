
#include <hip/hip_runtime.h>
#define imageChannels 1

__global__ void gpu2 (const unsigned char * InputImageData, const float * kernel, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
  __shared__ unsigned char tile[32][32][imageChannels];
  for (int k = 0; k < channels; ++k) {
    for (int m = 0; m < kernelSizeY; ++m) {
      for (int n = 0; n < kernelSizeX; ++n) {
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if( yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          tile[tx + m][ty + n][k] = InputImageData[(yIndex * dataSizeX + xIndex) * channels + k];
        } else {
          tile[tx + m][ty + n][k] = 0;
        }
      }
    }
  }
  __syncthreads();
  for (int k = 0; k < channels; ++k) {
    float sum = 0.0f;
    for (int m = 0; m < kernelSizeY; ++m) {
      int mm = kernelSizeY - 1 - m;
      for (int n = 0; n < kernelSizeX; ++n) {
        int nn = kernelSizeX - 1 - n;
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          sum += tile[ty + m][tx + n][k] * kernel[kernelSizeX * mm + nn];
        }
      }
    }
    outputImageData[(i * dataSizeX + j) * channels + k] = sum;
  }
}