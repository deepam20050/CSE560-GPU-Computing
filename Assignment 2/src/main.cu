#include "hip/hip_runtime.h"
/* Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>

#define kernelWidth 3
#define kernelHeight 3
#define imageChannels 1

using namespace std;

__constant__ float imageKernel_c[kernelHeight * kernelWidth];

void sequentialConvolution(const unsigned char*inputImageData, const float *kernel ,unsigned char * outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels)
{
	int i, j, m, n, mm, nn;
	int kCenterX, kCenterY;
	float sum;
	int yIndex, xIndex;

	kCenterX = kernelSizeX / 2;
	kCenterY = kernelSizeY / 2;

	for (int k=0; k<channels; k++) 
	{
		for (i = 0; i < dataSizeY; ++i)
		{
			for (j = 0; j < dataSizeX; ++j)
			{
				sum = 0;
				for (m = 0; m < kernelSizeY; ++m)
				{
					mm = kernelSizeY - 1 - m;

					for (n = 0; n < kernelSizeX; ++n)
					{
						nn = kernelSizeX - 1 - n;

						yIndex = i + m - kCenterY;
						xIndex = j + n - kCenterX;

						if (yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX)
							sum += inputImageData[(dataSizeX * yIndex + xIndex)*channels + k] * kernel[kernelSizeX * mm + nn];
					}
				}
				outputImageData[(dataSizeX * i + j)*channels + k] = sum;
			}
		}
	}
}

__global__ void gpu1 (const unsigned char * InputImageData, const float * kernel, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < dataSizeY && j < dataSizeX) {
		int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
		for (int k = 0; k < channels; ++k) {
			float sum = 0.0f;
			for (int m = 0; m < kernelSizeY; ++m) {
				int mm = kernelSizeY - 1 - m;
				for (int n = 0; n < kernelSizeX; ++n) {
					int nn = kernelSizeX - 1 - n;
					int yIndex = i + m - kCenterY;
					int xIndex = j + n - kCenterX;
					if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
						sum += InputImageData[(yIndex * dataSizeX + xIndex) * channels + k] * kernel[kernelSizeX * mm + nn];
					}
				}
			}
			outputImageData[(i * dataSizeX + j) * channels + k] = sum;
		}
	}
}

__global__ void gpu2 (const unsigned char * InputImageData, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
  __shared__ unsigned char tile[32][32][imageChannels];
  for (int k = 0; k < channels; ++k) {
    for (int m = 0; m < kernelSizeY; ++m) {
      for (int n = 0; n < kernelSizeX; ++n) {
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if( yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          tile[ty + m][tx + n][k] = InputImageData[(yIndex * dataSizeX + xIndex) * channels + k];
        } else {
          tile[ty + m][tx + n][k] = 0;
        }
      }
    }
  }
  __syncthreads();
  for (int k = 0; k < channels; ++k) {
    float sum = 0.0f;
    for (int m = 0; m < kernelSizeY; ++m) {
      int mm = kernelSizeY - 1 - m;
      for (int n = 0; n < kernelSizeX; ++n) {
        int nn = kernelSizeX - 1 - n;
        int yIndex = i + m - kCenterY;
        int xIndex = j + n - kCenterX;
        if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
          sum += tile[ty + m][tx + n][k] * imageKernel_c[kernelSizeX * mm + nn];
        }
      }
    }
    outputImageData[(i * dataSizeX + j) * channels + k] = sum;
  }
}


// Bonus Implementation
__global__ void gpu_tex (hipTextureObject_t imageKernel_texture, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < dataSizeY && j < dataSizeX) {
		int kCenterX = kernelSizeX / 2, kCenterY = kernelSizeY / 2;
		for (int k = 0; k < channels; ++k) {
			float sum = 0.0f;
			for (int m = 0; m < kernelSizeY; ++m) {
				int mm = kernelSizeY - 1 - m;
				for (int n = 0; n < kernelSizeX; ++n) {
					int nn = kernelSizeX - 1 - n;
					int yIndex = i + m - kCenterY;
					int xIndex = j + n - kCenterX;
					if(yIndex >= 0 && yIndex < dataSizeY && xIndex >= 0 && xIndex < dataSizeX) {
						sum += tex2D < unsigned char > (imageKernel_texture, xIndex, yIndex) * imageKernel_c[kernelSizeX * mm + nn];
					}
				}
			}
			outputImageData[(i * dataSizeX + j) * channels + k] = sum;
		}
	}
}

int main(int argc, char* argv[]){
	if(argc < 3) {
		cout<<"Usage: "<<argv[0]<<" <image_in> <image_out>\n";
		return 0;
	}
	// Read input image on host
	int imageWidth, imageHeight, bpp;
	const unsigned char* image_in = stbi_load( argv[1], &imageWidth, &imageHeight, &bpp, imageChannels );
	if(bpp != 1 || imageChannels != 1) {
		cout<<"Input image must be 8 bits per pixel, and sigle channel (grayscale).\n";
		return 0;
	}
	cout << "Image size: " << imageHeight << " x " << imageWidth << std::endl; 
	
	// Allocate output image memory on host
	unsigned char *image_out = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));
	unsigned char *image_gpu1 = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));
	unsigned char *image_gpu2 = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));
	unsigned char *image_gpu_tex = (unsigned char*) malloc(imageWidth*imageHeight*sizeof(unsigned char));

	// Setup image convolution kernel on host
	float imageKernel[kernelHeight*kernelWidth];
	for(int i=0; i< kernelWidth*kernelHeight; i++){
		imageKernel[i] = 1.0/(kernelHeight*kernelWidth);
	}
	struct timespec start_cpu, end_cpu;
	float msecs_cpu;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start_cpu);
	// Perform image convolution
	sequentialConvolution(image_in, imageKernel, image_out, kernelWidth, 
	kernelHeight, imageWidth, imageHeight, imageChannels);
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end_cpu);
	msecs_cpu = 1000.0 * (end_cpu.tv_sec - start_cpu.tv_sec) + (end_cpu.tv_nsec - start_cpu.tv_nsec)/1000000.0;
	printf("[CPU] %f microseconds.\n", msecs_cpu * 1000.0);

	// Add cuda code here
	
	// allocate memory on device
	int size = imageWidth * imageHeight;
	unsigned char *device_image_gpu1, *device_image_gpu2, *image_in_gpu, *device_image_tex;
	float *imageKernel_gpu;
	hipMalloc((void **)&device_image_gpu1, size * sizeof(unsigned char));
	hipMalloc((void **)&device_image_gpu2, size * sizeof(unsigned char));
	hipMalloc((void **)&device_image_tex, size * sizeof(unsigned char));
	hipMalloc((void **)&image_in_gpu, size * sizeof(unsigned char));
	hipMalloc((void **)&imageKernel_gpu, kernelHeight * kernelWidth * sizeof(float));
	hipMemcpy(image_in_gpu, image_in, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(imageKernel_gpu, imageKernel, kernelHeight * kernelWidth * sizeof(float), hipMemcpyHostToDevice);

	// creating blocks and grids
	dim3 block(16, 16);
	dim3 grid((imageHeight + block.x - 1) / block.x, (imageWidth + block.y - 1) / block.y);

	// launching GPU1 kernel
	gpu1<<<grid, block>>>(image_in_gpu, imageKernel_gpu, device_image_gpu1, kernelWidth, kernelHeight, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();
	hipMemcpy(image_gpu1, device_image_gpu1, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	string gpu1_png(argv[2]);
	gpu1_png.pop_back(); gpu1_png.pop_back(); gpu1_png.pop_back(); gpu1_png.pop_back();
	gpu1_png += "-GPU1.png";
	stbi_write_png(gpu1_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu1, 0);
	
	// copying imageKernel to imageKernel_c(constant memory)
	hipMemcpyToSymbol(HIP_SYMBOL(imageKernel_c), imageKernel, kernelHeight * kernelWidth * sizeof(float));
	
	// launching GPU2 kernel
	gpu2<<<grid, block>>>(image_in_gpu, device_image_gpu2, kernelWidth, kernelHeight, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();
	hipMemcpy(image_gpu2, device_image_gpu2, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	string gpu2_png(argv[2]);
	gpu2_png.pop_back(); gpu2_png.pop_back(); gpu2_png.pop_back(); gpu2_png.pop_back();
	gpu2_png += "-GPU2.png";
	stbi_write_png(gpu2_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu2, 0);
	
	// Texture memory part
	hipChannelFormatDesc channel_description = hipCreateChannelDesc<unsigned char>();
	hipArray_t cuArray;
	hipMallocArray(&cuArray, &channel_description, imageWidth, imageHeight);
	const size_t spitch = imageWidth * sizeof(unsigned char);
	hipMemcpy2DToArray(cuArray, 0, 0, image_in, spitch, imageWidth * sizeof(unsigned char), imageHeight, hipMemcpyHostToDevice);
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.readMode = hipReadModeElementType;
	hipTextureObject_t imageKernel_texture = 0;
	hipCreateTextureObject(&imageKernel_texture, &resDesc, &texDesc, NULL);
	// launching gpu_tex
	gpu_tex<<<grid, block>>>(imageKernel_texture, device_image_tex, kernelWidth, kernelHeight, imageWidth, imageHeight, imageChannels);
	hipDeviceSynchronize();
	hipMemcpy(image_gpu_tex, device_image_tex, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	string gputex_png(argv[2]);
	gputex_png.pop_back(); gputex_png.pop_back(); gputex_png.pop_back(); gputex_png.pop_back();
	gputex_png += "-GPU_TEX.png";
	stbi_write_png(gputex_png.c_str(), imageWidth, imageHeight, imageChannels, image_gpu_tex, 0);
	
	//Deallocate memory
	free(image_out);
	free(image_gpu1);
	free(image_gpu2);
	free(image_gpu_tex);
	hipFree(device_image_gpu1);
	hipFree(device_image_gpu2);
	hipFree(device_image_tex);
	hipFree(image_in_gpu);
	hipFree(imageKernel_gpu);
	return 0;
}
