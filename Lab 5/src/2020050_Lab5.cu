/* Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel1(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
    float x, s, c, root;
    for (int j = 0; j < 1000; j ++){
        x = (float)i;
        s = sinf(x); 
        c = cosf(x);
        root = sqrtf(s*s+c*c);
    }
    a[i] = a[i] + 2*root;
}

 __global__ void kernel2(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
    float x, s, c, root;
    for (int j = 0; j < 1000; j ++){
        x = (float)i;
        s = sinf(x); 
        c = cosf(x);
        root = sqrtf(s*s+c*c);
    }
    a[i] = a[i] - root;
}
 
 float difference(float *a, float *b, int n) 
{
    float diff = 0;
    for (int i = 0; i < n; i++) {
        diff += fabs(a[i]-b[i]);
    }
    return diff;
}


int main(){

    int blockSize = 256, nStreams = 4;
    int n = 4 * 1024 * blockSize * nStreams;
    int streamSize = n / nStreams;
    int streamBytes = streamSize * sizeof(float);
    int bytes = n * sizeof(float);


    float *a, *d_a, *b, *d_b;
    hipHostMalloc((void**)&a, bytes, hipHostMallocDefault);      
    hipMalloc((void**)&d_a, bytes);
    hipHostMalloc((void**)&b, bytes, hipHostMallocDefault);      
    hipMalloc((void**)&d_b, bytes);

    float ms;
   
    // create events
    hipEvent_t startEvent, stopEvent, dummyEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&dummyEvent);

    // creating stream
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i)
        hipStreamCreate(&stream[i]);

    // baseline method - using cudaMemcpy and calling kernels on default stream.
    memset(a, 0, bytes);
    hipEventRecord(startEvent,0);
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    kernel1<<<n/blockSize, blockSize>>>(d_a, 0);
    kernel2<<<n/blockSize, blockSize>>>(d_a, 0);
    hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time (baseline) : %f\n", ms);




    // using stream
    memset(b, 0, bytes);
    hipEventRecord(startEvent,0);

    const int gridSize = streamSize / blockSize;
    for (int i = 0; i < nStreams; ++i) {
        hipMemcpyAsync(d_b + i * streamSize, b + i * streamSize, streamBytes, hipMemcpyHostToDevice, stream[i]);
        kernel1<<<gridSize, blockSize, 0, stream[i]>>>(d_b, i * streamSize);
        kernel2<<<gridSize, blockSize, 0, stream[i]>>>(d_b, i * streamSize);
        hipMemcpyAsync(b + i * streamSize, d_b + i * streamSize, streamBytes, hipMemcpyDeviceToHost, stream[i]);
    }

    hipDeviceSynchronize();
    // TODO : (You can refer to the lab document for the syntax or use CUDA programming guide)
    // 1. [DONE] Use cudaMemcpyAsync to copy data from host to device
    // 2. [DONE] Call the kernels on non-default streams.
    // 3. [DONE] Use cudaMemcpyAsync to copy data back from device to host
    // 4. [DONE] Some hints - You can divide the complete array into chunks (you can use a for loop for calling kernels for every chunk) where different streams can evaluate for different chunks of the array. You can also call different kernels on different streams. 
    // To obtain marks, you only have to show that using streams, you get some improvement in time over the basic model of using only default stream. 
    // If you would like, you can visualize the memory copies and kernel calls using Nvidia Visual Profiler tool to see how different kernels are running on different streams.

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);

    // TODO : [DONE] Mention this time and difference between array a and b in the report. 
    // If your implementation is correct, this time will be slightly less than the baseline method and the difference between array a aand b will be zero.
    printf("Time : %f\n", ms);
    printf("  difference : %e\n", difference(a, b, n));

    // cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(dummyEvent);

    // [DONE] TODO : Destroy the streams. (You can refer to the lab document for the same)
    for (int i = 0; i < nStreams; i++) {
        hipStreamDestroy(stream[i]);
    }
      
    hipFree(d_a);
    hipHostFree(a);
    hipFree(d_b);
    hipHostFree(b);

    return 0;

}