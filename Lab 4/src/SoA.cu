/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <ctime>


#include <hip/hip_runtime.h>

#define ARRAY_SIZE 512

using namespace std;


struct SoA {
  int *keys;
  int *values;
};

__global__ void vector_add (SoA a, SoA b, SoA c) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < ARRAY_SIZE) {
    c.keys[i] = a.keys[i] + b.keys[i];
    c.values[i] = a.values[i] + b.values[i];
  }
}


int main(){
  struct SoA SoA_data1, SoA_data2, SoA_data3 ,d_SoA_data1, d_SoA_data2, d_SoA_data3;
  // malloc SoAdata1.keys, SoAdata1.values SoA_data2.keys, SoAdata3.keys etc
  SoA_data1.keys = (int *)malloc(ARRAY_SIZE * sizeof(int));
  SoA_data1.values = (int *)malloc(ARRAY_SIZE * sizeof(int));
  SoA_data2.keys = (int *)malloc(ARRAY_SIZE * sizeof(int));
  SoA_data2.values = (int *)malloc(ARRAY_SIZE * sizeof(int));
  SoA_data3.keys = (int *)malloc(ARRAY_SIZE * sizeof(int));
  SoA_data3.values = (int *)malloc(ARRAY_SIZE * sizeof(int));
  // initialize array keys, values
  for (int i = 0; i < ARRAY_SIZE; i++){
    SoA_data1.keys[i] = (i + 1);
    SoA_data1.values[i] = 2 * (i + 1);
    SoA_data2.keys[i] = 3 * (i + 1);
    SoA_data2.values[i] = 4 * (i + 1);
  }
  // cudaMalloc d_SoA_data1.keys, d_SoA_data1.values etc
  hipMalloc((void **) &d_SoA_data1.keys, ARRAY_SIZE * sizeof(int));
  hipMalloc((void **) &d_SoA_data1.values, ARRAY_SIZE * sizeof(int));
  hipMalloc((void **) &d_SoA_data2.keys, ARRAY_SIZE * sizeof(int));
  hipMalloc((void **) &d_SoA_data2.values, ARRAY_SIZE * sizeof(int));
  hipMalloc((void **) &d_SoA_data3.keys, ARRAY_SIZE * sizeof(int));
  hipMalloc((void **) &d_SoA_data3.values, ARRAY_SIZE * sizeof(int));
  // cudaMemcpy d_SoA_data1.keys, d_SoA_data1.values etc
	hipMemcpy(d_SoA_data1.keys, SoA_data1.keys, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_SoA_data1.values, SoA_data1.values, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_SoA_data2.keys, SoA_data2.keys, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_SoA_data2.values, SoA_data2.values, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
  // launching kernel and measuring time
  float gpu_elapsed = 0.0f;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start);
  vector_add<<<(ARRAY_SIZE/256)+1, 256>>>(d_SoA_data1, d_SoA_data2, d_SoA_data3);
  hipDeviceSynchronize();
  hipEventRecord(gpu_stop); 
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
	printf("[SoA] Time measured: %.10f milliseconds.\n", gpu_elapsed);
  // copy back to host array
  hipMemcpy(SoA_data3.keys, d_SoA_data3.keys, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(SoA_data3.values, d_SoA_data3.values, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  // Printing data
	// for (int i = 0; i < ARRAY_SIZE; ++i) {
	// 	printf("%d %d | %d %d | %d %d\n", SoA_data1.keys[i], SoA_data1.values[i], SoA_data2.keys[i], SoA_data2.values[i], SoA_data3.keys[i], SoA_data3.values[i]);
	// }
  // free the device memory
  hipFree(d_SoA_data1.keys);
  hipFree(d_SoA_data1.values);
  hipFree(d_SoA_data2.keys);
  hipFree(d_SoA_data2.values);
  hipFree(d_SoA_data3.keys);
  hipFree(d_SoA_data3.values);
  return 0;     
}