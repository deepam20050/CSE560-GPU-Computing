/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 512

using namespace std;

struct record {
  int key;
	int value;
};

__global__ void vector_add (record* a, record* b, record* c){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < ARRAY_SIZE) {
		c[i].key = a[i].key + b[i].key;
		c[i].value = a[i].value + b[i].value;
	}
}

int main() {
	struct record *AoS_data1, *AoS_data2, *AoS_data3 , *d_AoS_data1, *d_AoS_data2, *d_AoS_data3;
	// malloc AoSdata1, AoS_data2, AoSdata3
	AoS_data1 = (record *) malloc(ARRAY_SIZE * sizeof(record));
	AoS_data2 = (record *) malloc(ARRAY_SIZE * sizeof(record));
	AoS_data3 = (record *) malloc(ARRAY_SIZE * sizeof(record));
	// initialize array keys, values
	for (int i = 0; i < ARRAY_SIZE; i ++){
		AoS_data1[i].key = (i + 1);
		AoS_data1[i].value = 2 * (i + 1);
		AoS_data2[i].key = 3 * (i + 1);
		AoS_data2[i].value = 4 * (i + 1);
	}
	// cudaMalloc
	hipMalloc((void **) &d_AoS_data1, ARRAY_SIZE * sizeof(record));
	hipMalloc((void **) &d_AoS_data2, ARRAY_SIZE * sizeof(record));
	hipMalloc((void **) &d_AoS_data3, ARRAY_SIZE * sizeof(record));
	// cudaMemcpy
	hipMemcpy(d_AoS_data1, AoS_data1, ARRAY_SIZE * sizeof(record), hipMemcpyHostToDevice);
	hipMemcpy(d_AoS_data2, AoS_data2, ARRAY_SIZE * sizeof(record), hipMemcpyHostToDevice);
	// launching kernel and measuring time
	float gpu_elapsed = 0.0f;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start);
	vector_add<<<(ARRAY_SIZE/256)+1, 256>>>(d_AoS_data1, d_AoS_data2, d_AoS_data3);
	hipDeviceSynchronize();
	hipEventRecord(gpu_stop); 
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
	printf("[AoS] Time measured: %.10f milliseconds.\n", gpu_elapsed);
	// cudaMemcpy back to host array
	hipMemcpy(AoS_data3, d_AoS_data3, ARRAY_SIZE * sizeof(record), hipMemcpyDeviceToHost);
	// Printing data
	// for (int i = 0; i < ARRAY_SIZE; ++i) {
	// 	printf("%d %d | %d %d | %d %d\n", AoS_data1[i].key, AoS_data1[i].value, AoS_data2[i].key, AoS_data2[i].value, AoS_data3[i].key, AoS_data3[i].value);
	// }
	// free the device memory
	hipFree(d_AoS_data1);
	hipFree(d_AoS_data2);
	hipFree(d_AoS_data3);
	return 0;
}