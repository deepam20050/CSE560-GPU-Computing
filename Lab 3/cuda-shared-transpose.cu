#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 16

__global__ void matrixTransposeShared(float *d_out, float *d_in, int rows, int cols) {
    // 2D block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // 2D thread index within a block
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first element in the block
    int blockStart = bx * cols * BLOCK_SIZE + by * BLOCK_SIZE * rows;

    // Allocate shared memory to store elements of the block
    __shared__ float block[BLOCK_SIZE][BLOCK_SIZE];

    // Load the elements of the block into shared memory
    int row = blockStart + ty * rows + tx;
    if (row < rows * cols) {
        block[ty][tx] = d_in[row];
    }

    // Synchronize threads within the block
    __syncthreads();

    // Write the transposed block back to the output matrix
    int col = blockStart + ty * rows + tx;
    if (col < rows * cols) {
        d_out[col] = block[tx][ty];
    }
}

int main(int argc, char *argv[]) {
    float *d_in, *d_out;
    float h_A[4][5] = {{1, 2, 3, 4, 20}, {5, 6, 7, 8, 30}, {9, 10, 11, 12, 40}, {13, 14, 15, 16, 50}};

    int rows = 4;
    int cols = 5;

    // Allocate memory for the input and output matrices on the device
    hipMalloc((void **)&d_in, rows * cols * sizeof(float));
    hipMalloc((void **)&d_out, rows * cols * sizeof(float));

    // Copy the input matrix to the device
    hipMemcpy(d_in, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch the matrix transpose kernel
    dim3 grid((rows + BLOCK_SIZE - 1) / BLOCK_SIZE, (cols + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    matrixTransposeShared<<<grid, block>>>(d_out, d_in, rows, cols);
    float h_AT[5][4];
    // Copy the output matrix back to the host
    hipMemcpy(h_AT, d_out, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Print the transposed matrix
    for (int i = 0; i < cols; i++)
    {
        for (int j = 0; j < rows; j++)
        {
            std::cout << h_AT[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);
}
