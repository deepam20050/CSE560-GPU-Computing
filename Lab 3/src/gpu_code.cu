/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */
#include <hip/hip_runtime.h>
#include <cstdio>

const int ROWS = 1024;
const int COLS = 512;
const int THREADS = 16;

float A[ROWS][COLS], A_T[COLS][ROWS], A_T_A[COLS][COLS];

__global__ void global_transpose (float *d_A, float *d_A_T, int rows, int cols) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < rows && j < cols) {
    int idx_A = i * cols + j;
    int idx_A_T = j * rows + i;
    d_A_T[idx_A_T] = d_A[idx_A];
  }
}

__global__ void global_matmul (float *d_C, float *d_A, float *d_B, int rowsA, int colsA, int colsB) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < rowsA && j < colsB) {
    float sum = 0.0f;
    for (int k = 0; k < colsA; k++) {
      int idxA = i * colsA + k;
      int idxB = k * colsB + j;
      sum += d_A[idxA] * d_B[idxB];
    }
    int idxC = i * colsB + j;
    d_C[idxC] = sum;
  }
}

__global__ void shared_transpose (float *d_A, float *d_A_T, int rows, int cols) {
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int blockStart = bx * cols * THREADS + by * THREADS * rows;
  __shared__ float block[THREADS][THREADS];
  int row = blockStart + ty * rows + tx;
  if (row < rows * cols) {
      block[ty][tx] = d_A[row];
  }
  __syncthreads();
  int col = blockStart + ty * rows + tx;
  if (col < rows * cols) {
    d_A_T[col] = block[tx][ty];
  }
}

// Ref: Lecture 6, 04CUDA_Memories.pdf, Slides 41-42
__global__ void shared_matmul (float *a, float *b, float *ab, int width) {
  int tx = threadIdx.x, ty = threadIdx.y;
  int bx = blockIdx.x, by = blockIdx.y;
  __shared__ float s_a[THREADS][THREADS];
  __shared__ float s_b[THREADS][THREADS];
  int row = by * blockDim.y + ty;
  int col = bx * blockDim.x + tx;
  float result = 0;
  for(int p = 0; p < width/THREADS; ++p) {
    s_a[ty][tx] = a[row*width + (p*THREADS + tx)];
    s_b[ty][tx] = b[(p*THREADS + ty)*width + col];
    __syncthreads();
    for(int k = 0; k < THREADS; ++k)
      result += s_a[ty][k] * s_b[k][tx];
    __syncthreads();
  }
  ab[row * width + col] = result;
}

int main() {
  for (int i = 0; i < ROWS; ++i) {
    for (int j = 0; j < COLS; ++j) {
      A[i][j] = static_cast<float>((i + 1) * (j + 1));
    }
  }
  float *d_A, *d_A_T, *d_s_A_T, *d_A_T_A, *d_s_A_T_A;
  hipMalloc((void **)&d_A, ROWS * COLS * sizeof(float));
  hipMalloc((void **)&d_A_T, ROWS * COLS * sizeof(float));
  hipMalloc((void **)&d_s_A_T, ROWS * COLS * sizeof(float));
  hipMalloc((void **)&d_A_T_A, COLS * COLS * sizeof(float));
  hipMalloc((void **)&d_s_A_T_A, COLS * COLS * sizeof(float));
  hipMemcpy(d_A, A, ROWS * COLS * sizeof(float), hipMemcpyHostToDevice);
  // global memory code
  dim3 blockGlobalDim(THREADS, THREADS);
  dim3 gridGlobalDim((ROWS + blockGlobalDim.x - 1) / blockGlobalDim.x, (COLS + blockGlobalDim.y - 1) / blockGlobalDim.y);
  float gpu_elapsed = 0.0f;
  hipEvent_t gpu_start, gpu_stop;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start);
  global_transpose<<<gridGlobalDim, blockGlobalDim>>>(d_A, d_A_T, ROWS, COLS);
  hipDeviceSynchronize();
  hipEventRecord(gpu_stop); 
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
  printf("[GPU] Global Memory Transpose Time measured: %.9lf milliseconds.\n", gpu_elapsed);
  hipMemcpy(A_T, d_A_T, ROWS * COLS * sizeof(float), hipMemcpyDeviceToHost);
  gpu_elapsed = 0.0f;
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start);
  global_matmul<<<gridGlobalDim, blockGlobalDim>>>(d_A_T_A, d_A_T, d_A, COLS, ROWS, ROWS);
  hipDeviceSynchronize();
  hipEventRecord(gpu_stop); 
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
  printf("[GPU] Global Memory A_T * A Time measured: %.9lf milliseconds.\n", gpu_elapsed);
  hipMemcpy(A_T_A, d_A_T_A, COLS * COLS * sizeof(float), hipMemcpyDeviceToHost);
  // shared memory code
  dim3 sharedGridDim((ROWS + THREADS - 1) / THREADS, (COLS + THREADS - 1) / THREADS);
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start);
  shared_transpose<<<sharedGridDim, blockGlobalDim>>>(d_A, d_s_A_T, ROWS, COLS);
  hipDeviceSynchronize();
  hipEventRecord(gpu_stop); 
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
  printf("[GPU] Shared Memory Transpose Time measured: %.9lf milliseconds.\n", gpu_elapsed);
  hipMemcpy(A_T, d_s_A_T, ROWS * COLS * sizeof(float), hipMemcpyDeviceToHost);
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);
  hipEventRecord(gpu_start);
  shared_matmul<<<sharedGridDim, blockGlobalDim>>>(d_s_A_T, d_A, d_s_A_T_A, COLS);
  hipDeviceSynchronize();
  hipEventRecord(gpu_stop); 
  hipEventSynchronize(gpu_stop);
  hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
  printf("[GPU] Shared Memory A_T * A Time measured: %.9lf milliseconds.\n", gpu_elapsed);
  hipMemcpy(A_T_A, d_s_A_T_A, COLS * COLS * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_A_T);
  hipFree(d_s_A_T);
  hipFree(d_A_T_A);
  hipFree(d_s_A_T_A);
  return 0;
}