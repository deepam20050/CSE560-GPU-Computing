#include <hip/hip_runtime.h>
#include <iostream>

__global__ void transpose(float *d_A, float *d_AT, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols)
    {
        int index_A = i * cols + j;
        int index_AT = j * rows + i;

        d_AT[index_AT] = d_A[index_A];
    }
}

int main()
{
    // Input matrix
    float h_A[4][5] = {{1, 2, 3, 4, 20}, {5, 6, 7, 8, 30}, {9, 10, 11, 12, 40}, {13, 14, 15, 16, 50}};

    int rows = 4;
    int cols = 5;

    // Allocate device memory
    float *d_A, *d_AT;
    hipMalloc((void **)&d_A, rows * cols * sizeof(float));
    hipMalloc((void **)&d_AT, rows * cols * sizeof(float));

    // Copy input matrix to device memory
    hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch transpose kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
    transpose<<<gridDim, blockDim>>>(d_A, d_AT, rows, cols);

    // Copy result back to host memory
    float h_AT[5][4];
    hipMemcpy(h_AT, d_AT, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < cols; i++)
    {
        for (int j = 0; j < rows; j++)
        {
            std::cout << h_AT[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_AT);

    return 0;
}
