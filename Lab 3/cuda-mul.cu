#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matmul(float *d_C, float *d_A, float *d_B, int rowsA, int colsA, int colsB)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rowsA && j < colsB)
    {
        float sum = 0.0f;
        for (int k = 0; k < colsA; k++)
        {
            int indexA = i * colsA + k;
            int indexB = k * colsB + j;

            sum += d_A[indexA] * d_B[indexB];
        }
        int indexC = i * colsB + j;
        d_C[indexC] = sum;
    }
}

int main()
{
    // Input matrices
    float h_A[4][3] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}, {10, 11, 12}};
    float h_B[3][4] = {{13, 14, 15, 16}, {17, 18, 19, 20}, {21, 22, 23, 24}};

    int rowsA = 4;
    int colsA = 3;
    int colsB = 4;

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, rowsA * colsA * sizeof(float));
    hipMalloc((void **)&d_B, colsA * colsB * sizeof(float));
    hipMalloc((void **)&d_C, rowsA * colsB * sizeof(float));

    // Copy input matrices to device memory
    hipMemcpy(d_A, h_A, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, colsA * colsB * sizeof(float), hipMemcpyHostToDevice);

    // Launch matrix multiplication kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((rowsA + blockDim.x - 1) / blockDim.x, (colsB + blockDim.y - 1) / blockDim.y);
    matmul<<<gridDim, blockDim>>>(d_C, d_A, d_B, rowsA, colsA, colsB);

    // Copy result back to host memory
    float h_C[4][4];
    hipMemcpy(h_C, d_C, rowsA * colsB * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < colsB; j++) {
          std::cout << h_C[i][j] << " "; 
        }
        std::cout << '\n';
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
