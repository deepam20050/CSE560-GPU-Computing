#include "hip/hip_runtime.h"
/**
Copyright (c) 2023 [Ojaswa Sharma, IIIT Delhi]. All rights reserved.

This computer code is proprietary and confidential. It is provided for the sole use of students enrolled in CSE 560 - GPU Computing. Any unauthorized use, reproduction, distribution or modification of this code, in whole or in part, is strictly prohibited.

By accessing and using the code, you agree to be bound by the terms and conditions of this notice. Unauthorized use may result in severe civil and criminal penalties, and will be prosecuted to the maximum extent possible under the law.

This notice constitutes an agreement between you and the author of the code, and may only be modified in writing signed by both parties.
**/

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

/*
 References:
 [1] http://stackoverflow.com/questions/23711681/generating-custom-color-palette-for-julia-set
 [2] http://www.cs.rit.edu/~ncs/color/t_convert.html
*/

#include <stdio.h>
#include <stdlib.h>
#include <complex>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

#define N 1024
#define SQRT_2 1.4142
#define MAX_ITER 512

void HSVtoRGB( float *r, float *g, float *b, float h, float s, float v );
void saveImage(int width, int height, unsigned char * bitmap, complex<float> seed);
void compute_julia_CPU(complex<float> c, unsigned char * image);
void compute_julia_GPU(complex<float> c, unsigned char * image);
bool compare_CPU_GPU(unsigned char *image_CPU, unsigned char *image_GPU);


__global__ void dummy_kernel()
{
	int tx = threadIdx.x + blockDim.x*blockIdx.x;
	tx++;
}

int main(int argc, char **argv)
{
	complex<float> c(0.285f, 0.01f);
	if(argc > 2)
	{
		c.real(atof(argv[1]));
		c.imag(atof(argv[2]));
	} else
		fprintf(stderr, "Usage: %s <real> <imag>\nWhere <real> and <imag> form the complex seed for the Julia set.\n", argv[0]);

	unsigned char *image_CPU_host = new unsigned char[N*N*3]; //RGB image
	unsigned char *image_GPU_host = new unsigned char[N*N*3]; //RGB image

	// Compute Julia set on CPU
	struct timespec start_cpu, end_cpu;
	float msecs_cpu;
	fprintf(stderr, "Performing Julia set computation on CPU... ");	
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start_cpu);	

	compute_julia_CPU(c, image_CPU_host);

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end_cpu);
	msecs_cpu = 1000.0 * (end_cpu.tv_sec - start_cpu.tv_sec) + (end_cpu.tv_nsec - start_cpu.tv_nsec)/1000000.0;
	fprintf(stderr, "done in %f milliseconds.\n", msecs_cpu);

	// Compute Julia set on GPU
	hipEvent_t start_gpu, end_gpu;
	float msecs_gpu;
	fprintf(stderr, "Performing Julia set computation on GPU... ");	
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);

	compute_julia_GPU(c, image_CPU_host);

	hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);
	hipEventElapsedTime(&msecs_gpu, start_gpu, end_gpu);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);
	fprintf(stderr, "done in %f milliseconds.\n", msecs_gpu);

	bool result = compare_CPU_GPU(image_CPU_host, image_GPU_host);
	fprintf(stderr, "CPU-GPU results do %smatch!\n", (result)?"":"not ");

	saveImage(N, N, image_CPU_host, c);
	delete[] image_CPU_host;
	delete[] image_GPU_host;
}

void compute_julia_CPU(complex<float> c, unsigned char * image)
{
	complex<float> z_old(0.0f, 0.0f);
	complex<float> z_new(0.0f, 0.0f);
	for(int y=0; y<N; y++)
		for(int x=0; x<N; x++)
		{
			z_new.real(4.0f * x / (N) - 2.0f);
			z_new.imag(4.0f * y / (N) - 2.0f);
			int i;
			for(i=0; i<MAX_ITER; i++)
			{
				z_old.real(z_new.real());
				z_old.imag(z_new.imag());
				z_new = pow(z_new, 2);
				z_new += c;
				if(norm(z_new) > 4.0f) break;
			}
			float brightness = (i<MAX_ITER) ? 1.0f : 0.0f;
			float hue = (i % MAX_ITER)/float(MAX_ITER - 1);
			hue = (120*sqrtf(hue) + 150);
			float r, g, b;
			HSVtoRGB(&r, &g, &b, hue, 1.0f, brightness);
			image[(x + y*N)*3 + 0] = (unsigned char)(b*255);
			image[(x + y*N)*3 + 1] = (unsigned char)(g*255);
			image[(x + y*N)*3 + 2] = (unsigned char)(r*255);
		}
}

void compute_julia_GPU(complex<float> c, unsigned char * image) {
	//TODO: Implement Julia set computation here. Remove the two lines below.
	dummy_kernel<<<1024, 1024>>>();
	hipDeviceSynchronize();
}

//Returns true if GPU results match CPU results, else returns false
bool compare_CPU_GPU(unsigned char *image_CPU, unsigned char *image_GPU)
{
	bool result = true;
	int nelem = N*N*3;
	for (int i=0; i<nelem; i++) {
		if (image_CPU[i] != image_GPU[i]) {result = false; break;}
	}

	return result;
}

void saveImage(int width, int height, unsigned char * bitmap, complex<float> seed)
{
	char imageName[256];
	sprintf(imageName, "Julia %.3f + i%.3f.png", seed.real(), seed.imag());
	stbi_write_png(imageName, width, height, 3, bitmap, width*3);
	fprintf(stderr, "Image saved as: %s\n", imageName);
}

// r,g,b values are from 0 to 1
// h = [0,360], s = [0,1], v = [0,1]
//		if s == 0, then h = -1 (undefined)
void HSVtoRGB( float *r, float *g, float *b, float h, float s, float v )
{
	int i;
	float f, p, q, t;
	if( s == 0 ) {
		// achromatic (grey)
		*r = *g = *b = v;
		return;
	}
	h /= 60;			// sector 0 to 5
	i = floor( h );
	f = h - i;			// factorial part of h
	p = v * ( 1 - s );
	q = v * ( 1 - s * f );
	t = v * ( 1 - s * ( 1 - f ) );
	switch( i ) {
		case 0:
			*r = v;
			*g = t;
			*b = p;
			break;
		case 1:
			*r = q;
			*g = v;
			*b = p;
			break;
		case 2:
			*r = p;
			*g = v;
			*b = t;
			break;
		case 3:
			*r = p;
			*g = q;
			*b = v;
			break;
		case 4:
			*r = t;
			*g = p;
			*b = v;
			break;
		default:		// case 5:
			*r = v;
			*g = p;
			*b = q;
			break;
	}
}
