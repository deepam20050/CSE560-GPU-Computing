#include <hip/hip_runtime.h>

__global__ void compute_julia_GPU_optimized(float *d_output, int width, int height, float real_min, float real_max, float imag_min, float imag_max, int max_iter) {

    // block size
    const int block_size = 32;
    // block id and thread id
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // index of the output array
    int index = bx * block_size * height + by * block_size + ty * width + tx;
    // coordinate in the complex plane
    float real = real_min + (float)(tx + bx * block_size) * (real_max - real_min) / (float)(width - 1);
    float imag = imag_min + (float)(ty + by * block_size) * (imag_max - imag_min) / (float)(height - 1);
    // store values in shared memory
    __shared__ float s_real[block_size][block_size];
    __shared__ float s_imag[block_size][block_size];
    s_real[tx][ty] = real;
    s_imag[tx][ty] = imag;
    __syncthreads();
    // calculate the number of iterations for each pixel
    float c_real = -0.7f;
    float c_imag = 0.27015f;
    float z_real = real;
    float z_imag = imag;
    int iter = 0;
    for (; iter < max_iter; ++iter) {
        float z_real_squared = z_real * z_real;
        float z_imag_squared = z_imag * z_imag;
        if (z_real_squared + z_imag_squared > 4.0f) {
            break;
        }
        z_imag = 2.0f * z_real * z_imag + c_imag;
        z_real = z_real_squared - z_imag_squared + c_real;
    }
    d_output[index] = (float)iter / (float)max_iter;
}
