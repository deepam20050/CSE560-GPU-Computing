#include <hip/hip_runtime.h>
#include <iostream>

__global__ void julia_set_kernel(int *output, float *x, float *y, int width, int height, float real_c, float imag_c)
{
    __shared__ float s_x[16][16];
    __shared__ float s_y[16][16];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * width;

    if (i >= width || j >= height) return;

    s_x[threadIdx.y][threadIdx.x] = x[i];
    s_y[threadIdx.y][threadIdx.x] = y[j];

    __syncthreads();

    float real = s_x[threadIdx.y][threadIdx.x];
    float imag = s_y[threadIdx.y][threadIdx.x];
    int value = 0;

    for (int iter = 0; iter < 255; iter++)
    {
        float r2 = real * real;
        float i2 = imag * imag;

        if (r2 + i2 > 4.0f)
        {
            value = iter;
            break;
        }

        imag = 2.0f * real * imag + imag_c;
        real = r2 - i2 + real_c;
    }

    output[index] = value;
}

int main()
{
    int width = 800, height = 600;
    int size = width * height;
    float real_c = -0.8f, imag_c = 0.156f;
    float *x, *y;
    int *output;

    hipMallocManaged(&x, width * sizeof(float));
    hipMallocManaged(&y, height * sizeof(float));
    hipMallocManaged(&output, size * sizeof(int));

    for (int i = 0; i < width; i++)
    {
        x[i] = (i / (float)width) * 4.0f - 2.0f;
    }

    for (int i = 0; i < height; i++)
    {
        y[i] = (i / (float)height) * 4.0f - 2.0f;
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    julia_set_kernel<<<gridSize, blockSize>>>(output, x, y, width, height, real_c, imag_c);

    hipDeviceSynchronize();

    // Convert the output array to color values and save the image

    hipFree(x);
    hipFree(y);
    hipFree(output);

    return 0;
}
