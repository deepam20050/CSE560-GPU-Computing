#include "hip/hip_runtime.h"
#define BLOCK_SIZE 1024

__global__ void compute_julia_GPU_kernel(complex<float> c, unsigned char *image, int width, int height)
{
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    int gy = blockIdx.y * blockDim.y + threadIdx.y;

    if (gx >= width || gy >= height)
        return;

    int i = gx + gy * width;

    complex<float> z(gx * 4.0f / width - 2.0f, gy * 4.0f / height - 2.0f);
    int iterations = 0;
    while (abs(z) <= SQRT_2 && iterations < MAX_ITER)
    {
        z = z * z + c;
        iterations++;
    }

    float r, g, b;
    HSVtoRGB(&r, &g, &b, (float)iterations / MAX_ITER, 1.0f, 1.0f);

    image[3 * i] = (unsigned char)(255.0f * r);
    image[3 * i + 1] = (unsigned char)(255.0f * g);
    image[3 * i + 2] = (unsigned char)(255.0f * b);
}

void compute_julia_GPU(complex<float> c, unsigned char *image)
{
    int width = N;
    int height = N;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    compute_julia_GPU_kernel<<<gridDim, blockDim>>>(c, image, width, height);
}
