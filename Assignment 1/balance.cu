#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SIZE 256

__global__ void balanceLoadKernel(int *workLoad, int *threadLoad)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    int loadPerThread = (workLoad[0] + numThreads - 1) / numThreads;
    int start = tid * loadPerThread;
    int end = start + loadPerThread;
    if (end > workLoad[0])
    {
        end = workLoad[0];
    }

    int myLoad = end - start;
    threadLoad[tid] = myLoad;
}

int main()
{
    int workLoad = 100000;
    int *d_workLoad;
    int *d_threadLoad;
    int *threadLoad = (int *)malloc(sizeof(int) * BLOCK_SIZE * gridDim.x);

    hipMalloc((void **)&d_workLoad, sizeof(int));
    hipMalloc((void **)&d_threadLoad, sizeof(int) * BLOCK_SIZE * gridDim.x);

    hipMemcpy(d_workLoad, &workLoad, sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = ceil((float)BLOCK_SIZE / (float)gridDim.x);

    balanceLoadKernel<<<numBlocks, BLOCK_SIZE>>>(d_workLoad, d_threadLoad);
    hipDeviceSynchronize();
    hipMemcpy(threadLoad, d_threadLoad, sizeof(int) * BLOCK_SIZE * gridDim.x, hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_SIZE * gridDim.x; i++)
    {
        std::cout << "Thread " << i << ": " << threadLoad[i] << std::endl;
    }

    hipFree(d_workLoad);
    hipFree(d_threadLoad);
    free(threadLoad);

    return 0;
}
