#include "hip/hip_runtime.h"
/**
Copyright (c) 2023 [Ojaswa Sharma, IIIT Delhi]. All rights reserved.

This computer code is proprietary and confidential. It is provided for the sole use of students enrolled in CSE 560 - GPU Computing. Any unauthorized use, reproduction, distribution or modification of this code, in whole or in part, is strictly prohibited.

By accessing and using the code, you agree to be bound by the terms and conditions of this notice. Unauthorized use may result in severe civil and criminal penalties, and will be prosecuted to the maximum extent possible under the law.

This notice constitutes an agreement between you and the author of the code, and may only be modified in writing signed by both parties.
**/

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */

/*
 References:
 [1] http://stackoverflow.com/questions/23711681/generating-custom-color-palette-for-julia-set
 [2] http://www.cs.rit.edu/~ncs/color/t_convert.html
 [3] https://thrust.github.io/doc/group__complex__numbers.html
*/

#include <cstdio>
#include <cstdlib>
#include <complex>
#include <cstring>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

const int N = 1024;
const float SQRT_2 = 1.4142;
const int MAX_ITER = 512;

__host__ __device__ void HSVtoRGB( float *r, float *g, float *b, float h, float s, float v );
void saveImage(int width, int height, unsigned char * bitmap, complex<float> seed, int who);
void compute_julia_CPU(complex<float> c, unsigned char * image);
void compute_julia_GPU(complex<float> c, unsigned char * image);
bool compare_CPU_GPU(unsigned char *image_CPU, unsigned char *image_GPU);

__global__ void compute_julia_kernel(unsigned char *image, thrust::complex<float> c) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int index = y * N + x;
	if (x < N && y < N) {
		thrust::complex<float> z_old(0.0f, 0.0f);
		thrust::complex<float> z_new(4.0f * x / N - 2.0f, 4.0f * y / N - 2.0f);
		int iter = 0;
		while (iter < MAX_ITER) {
			z_old.real(z_new.real());
			z_old.imag(z_new.imag());
			z_new *= z_new;
			z_new += c;
			if (thrust::norm(z_new) > 4.0f) {
				break;
			}
			++iter;
		}
		float brightness = (iter < MAX_ITER) ? 1.0f : 0.0f;
		float hue = (iter % MAX_ITER) / float(MAX_ITER - 1);
		hue = (120 * sqrtf(hue) + 150);
		float r, g, b;
		HSVtoRGB(&r, &g, &b, hue, 1.0f, brightness);
		image[index * 3] = (unsigned char)(b * 255);
		image[index * 3 + 1] = (unsigned char)(g * 255);
		image[index * 3 + 2] = (unsigned char)(r * 255);
	}
}

__global__ void dummy_kernel()
{
	int tx = threadIdx.x + blockDim.x*blockIdx.x;
	tx++;
}

int main(int argc, char **argv)
{
	complex<float> c(0.285f, 0.01f);
	if(argc > 2)
	{
		c.real(atof(argv[1]));
		c.imag(atof(argv[2]));
	} else
		fprintf(stderr, "Usage: %s <real> <imag>\nWhere <real> and <imag> form the complex seed for the Julia set.\n", argv[0]);

	unsigned char *image_CPU_host = new unsigned char[N*N*3]; //RGB image
	unsigned char *image_GPU_host = new unsigned char[N*N*3]; //RGB image

	// Compute Julia set on CPU
	struct timespec start_cpu, end_cpu;
	float msecs_cpu;
	fprintf(stderr, "Performing Julia set computation on CPU... ");	
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start_cpu);	

	compute_julia_CPU(c, image_CPU_host);

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end_cpu);
	msecs_cpu = 1000.0 * (end_cpu.tv_sec - start_cpu.tv_sec) + (end_cpu.tv_nsec - start_cpu.tv_nsec)/1000000.0;
	fprintf(stderr, "done in %f milliseconds.\n", msecs_cpu);

	// Compute Julia set on GPU
	hipEvent_t start_gpu, end_gpu;
	float msecs_gpu;
	fprintf(stderr, "Performing Julia set computation on GPU... ");	
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);

	compute_julia_GPU(c, image_GPU_host);

	hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);
	hipEventElapsedTime(&msecs_gpu, start_gpu, end_gpu);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);
	fprintf(stderr, "done in %f milliseconds.\n", msecs_gpu);

	bool result = compare_CPU_GPU(image_CPU_host, image_GPU_host);
	fprintf(stderr, "CPU-GPU results do %smatch!\n", (result)?"":"not ");

	saveImage(N, N, image_CPU_host, c, 0); // for CPU image
	saveImage(N, N, image_GPU_host, c, 1); // for GPU image
	delete[] image_CPU_host;
	delete[] image_GPU_host;
}

void compute_julia_CPU(complex<float> c, unsigned char * image)
{
	complex<float> z_old(0.0f, 0.0f);
	complex<float> z_new(0.0f, 0.0f);
	for(int y=0; y<N; y++)
		for(int x=0; x<N; x++)
		{
			z_new.real(4.0f * x / (N) - 2.0f);
			z_new.imag(4.0f * y / (N) - 2.0f);
			int i;
			for(i=0; i<MAX_ITER; i++)
			{
				z_old.real(z_new.real());
				z_old.imag(z_new.imag());
				z_new = pow(z_new, 2);
				z_new += c;
				if(norm(z_new) > 4.0f) break;
			}
			float brightness = (i<MAX_ITER) ? 1.0f : 0.0f;
			float hue = (i % MAX_ITER)/float(MAX_ITER - 1);
			hue = (120*sqrtf(hue) + 150);
			float r, g, b;
			HSVtoRGB(&r, &g, &b, hue, 1.0f, brightness);
			image[(x + y*N)*3 + 0] = (unsigned char)(b*255);
			image[(x + y*N)*3 + 1] = (unsigned char)(g*255);
			image[(x + y*N)*3 + 2] = (unsigned char)(r*255);
		}
}

void compute_julia_GPU(complex<float> c, unsigned char * image) {
	int size = N * N * 3;
	unsigned char *device_image;
	hipMalloc((void **)&device_image, size);
	dim3 block(32, 32);
	dim3 grid(N / block.x, N / block.y);
	thrust::complex <float> c_thrust(c.real(), c.imag()); 
	compute_julia_kernel<<<grid, block>>>(device_image, c_thrust);
	hipDeviceSynchronize();
	hipMemcpy(image, device_image, size, hipMemcpyDeviceToHost);
	hipFree(device_image);
}

//Returns true if GPU results match CPU results, else returns false
bool compare_CPU_GPU(unsigned char *image_CPU, unsigned char *image_GPU) {
	constexpr int nelem = N*N*3;
	float mean_cpu = 0.0f;
	for (int i = 0; i < nelem; ++i) {
		mean_cpu += image_CPU[i];
	}
	mean_cpu /= nelem;
	float sst = 0.0f, sse = 0.0f;
	for (int i = 0; i < nelem; ++i) {
		sst += (image_CPU[i] - mean_cpu) * (image_CPU[i] - mean_cpu);
		sse += (image_GPU[i] - mean_cpu) * (image_GPU[i] - mean_cpu);
	}
	return (sse / sst) >= 0.99f;
}

void saveImage(int width, int height, unsigned char * bitmap, complex<float> seed, int who)
{
	char imageName[256];
	sprintf(imageName, "%d %d Julia %.3f + i%.3f.png", who, N, seed.real(), seed.imag());
	stbi_write_png(imageName, width, height, 3, bitmap, width*3);
	fprintf(stderr, "Image saved as: %s\n", imageName);
}

// r,g,b values are from 0 to 1
// h = [0,360], s = [0,1], v = [0,1]
//		if s == 0, then h = -1 (undefined)
__host__ __device__ void HSVtoRGB( float *r, float *g, float *b, float h, float s, float v )
{
	int i;
	float f, p, q, t;
	if( s == 0 ) {
		// achromatic (grey)
		*r = *g = *b = v;
		return;
	}
	h /= 60;			// sector 0 to 5
	i = floor( h );
	f = h - i;			// factorial part of h
	p = v * ( 1 - s );
	q = v * ( 1 - s * f );
	t = v * ( 1 - s * ( 1 - f ) );
	switch( i ) {
		case 0:
			*r = v;
			*g = t;
			*b = p;
			break;
		case 1:
			*r = q;
			*g = v;
			*b = p;
			break;
		case 2:
			*r = p;
			*g = v;
			*b = t;
			break;
		case 3:
			*r = p;
			*g = q;
			*b = v;
			break;
		case 4:
			*r = t;
			*g = p;
			*b = v;
			break;
		default:		// case 5:
			*r = v;
			*g = p;
			*b = q;
			break;
	}
}
