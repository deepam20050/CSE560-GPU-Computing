#include "hip/hip_runtime.h"
void compute_julia_GPU(complex<float> c, unsigned char * image)
{
    int size = N * N * 3;
    unsigned char *device_image;
    hipMalloc((void **)&device_image, size);

    dim3 block(16, 16);
    dim3 grid(N / block.x, N / block.y);

    // Launch CUDA kernel to compute Julia set on GPU
    compute_julia_kernel<<<grid, block>>>(device_image, c);

    hipMemcpy(image, device_image, size, hipMemcpyDeviceToHost);

    hipFree(device_image);
}

__global__ void compute_julia_kernel(unsigned char *image, complex<float> c)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int index = y * N + x;

    if (x < N && y < N)
    {
        complex<float> z;
        z.real(SQRT_2 * (2.0f * x / N - 1.0f));
        z.imag(SQRT_2 * (2.0f * y / N - 1.0f));

        int iterations = 0;
        while (iterations < MAX_ITER && abs(z) < 2.0f)
        {
            z = z * z + c;
            iterations++;
        }

        float h = (float)iterations / MAX_ITER;
        float s = 1.0f;
        float v = 1.0f;
        float r, g, b;
        HSVtoRGB(&r, &g, &b, h, s, v);

        image[index * 3 + 0] = (unsigned char)(255.0f * r);
        image[index * 3 + 1] = (unsigned char)(255.0f * g);
        image[index * 3 + 2] = (unsigned char)(255.0f * b);
    }
}
