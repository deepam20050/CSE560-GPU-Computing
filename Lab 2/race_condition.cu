/*
 * Deepam Sarmah
 * 2020050
 * deepam20050@iiitd.ac.in
 */

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>


__global__ void kernel(int *a_d, int *max) {
  *a_d += 1;
  if (*a_d > *max) {
    *max = *a_d;
  }
}

int main() {
  int a = 0, maxVal = 0, *a_d, *max;
  
  hipMalloc((void**)&a_d, sizeof(int));
  hipMalloc((void**)&max, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(max, &maxVal, sizeof(int), hipMemcpyHostToDevice);

  float milliseconds = 0.0f;
  hipEvent_t start, stop;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  
  kernel<<<1000, 1000>>>(a_d, max);
  hipDeviceSynchronize();
  
  hipEventRecord(stop); 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&maxVal, max, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("a = %d\nmax = %d\n", a, maxVal);
  hipFree(a_d);
  hipFree(max);
  printf("Time measured by GPU: %.10f milliseconds.\n", milliseconds);
}